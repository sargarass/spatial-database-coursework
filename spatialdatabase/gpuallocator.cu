#include "gpuallocator.h"

gpudb::gpuAllocator &gpudb::gpuAllocator::getInstance() {
    static gpuAllocator *allocator = new gpuAllocator();
    static bool init = false;
    if (init == false) {
        init = true;
        SingletonFactory::getInstance().registration<gpuAllocator>(allocator);
        dynamic_cast<Singleton*>(allocator)->dependOn(Log::getInstance());
    }
    return *allocator;
}

bool gpudb::gpuAllocator::free(void *ptr) {
    if (ptr == nullptr) {
        return false;
    }

    if (!memoryPtrs.erase(reinterpret_cast<uintptr_t>(ptr))) {
        gLogWrite(LOG_MESSAGE_TYPE::DEBUG, "memory was not alloced");
        return false;
    }
    hipFree(ptr);
    return true;
}

void gpudb::gpuAllocator::freeAll() {
    gLogWrite(LOG_MESSAGE_TYPE::DEBUG, "freeing all memory");
    for (auto& ptr : memoryPtrs) {
        gLogWrite(LOG_MESSAGE_TYPE::DEBUG, "%p", ptr);
        hipFree((void*)ptr);
    }
    memoryPtrs.clear();
    hipDeviceReset();
}
