#include "testrequests.h"
#include "constobjects.h"
using namespace gpudb;

__device__
bool tester(gpudb::CRow const &row) {
    Date validS = row.getKeyValidTimeStart();
    Date validE = row.getKeyValidTimeEnd();
    return (validS.getYear() >= -21669);
}


__device__ Predicate h_tester = tester;
Predicate getTesterPointer() {
    Predicate p;
    hipMemcpyFromSymbol(&p, HIP_SYMBOL(h_tester), sizeof(Predicate));
    return p;
}
