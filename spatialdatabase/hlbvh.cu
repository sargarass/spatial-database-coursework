#include "hip/hip_runtime.h"
#include "hlbvh.h"
#include "../externlibs/cub/hipcub/hipcub.hpp"
bool gpudb::HLBVH::alloc(uint32_t size) {
    do {
        links = gpuAllocator::getInstance().alloc<int>( 2 * size );
        if (links == nullptr) { break; }
        parents = gpuAllocator::getInstance().alloc<int>( 2 * size );
        if (parents == nullptr) { break; }
        ranges = gpuAllocator::getInstance().alloc<uint2>( 2 * size );
        if (ranges == nullptr) { break; }
        aabbMin = gpuAllocator::getInstance().alloc<float4>( 2 * size );
        if (aabbMin == nullptr) { break; }
        aabbMax = gpuAllocator::getInstance().alloc<float4>( 2 * size );
        if (aabbMax == nullptr) { break; }

        this->numBVHLevels = 0;
        this->numNodes = 0;
        this->numReferences = size;
        return true;
    } while(0);

    free();
    return false;
}

void gpudb::HLBVH::free() {
    if (links) {
        gpuAllocator::getInstance().free(links);
    }

    if (parents) {
        gpuAllocator::getInstance().free(parents);
    }

    if (ranges) {
        gpuAllocator::getInstance().free(ranges);
    }

    if (aabbMin) {
        gpuAllocator::getInstance().free(aabbMin);
    }

    if (aabbMax) {
        gpuAllocator::getInstance().free(aabbMax);
    }
    this->numBVHLevels = 0;
    this->numNodes = 0;
    this->numReferences = 0;
}

static __device__ __inline__
uint getGlobalIdx3DZ() {
    uint blockId = blockIdx.x
                 + blockIdx.y * gridDim.x
                 + gridDim.x * gridDim.y * blockIdx.z;
    return blockId * blockDim.z + threadIdx.z;
}

static __device__ __inline__
uint getGlobalIdx3DZXY()
{
    uint blockId = blockIdx.x
             + blockIdx.y * gridDim.x
             + gridDim.x * gridDim.y * blockIdx.z;
    return blockId * (blockDim.x * blockDim.y * blockDim.z)
              + (threadIdx.z * (blockDim.x * blockDim.y))
              + (threadIdx.y * blockDim.x)
              + threadIdx.x;
}

static dim3 gridConfigure(uint64_t problemSize, dim3 block) {
    /// TODO
    /*dim3 MaxGridDim = {(uint)LibResouces::getCudaProperties(0).maxGridDimensionSize[0],
                       (uint)LibResouces::getCudaProperties(0).maxGridDimensionSize[1],
                       (uint)LibResouces::getCudaProperties(0).maxGridDimensionSize[2]};
    dim3 gridDim = {1, 1, 1};

    uint64_t blockSize = block.x * block.y * block.z;

    if (problemSize > MaxGridDim.y * MaxGridDim.x * blockSize) {
        gridDim.z = problemSize / MaxGridDim.x * MaxGridDim.y * blockSize;
        problemSize = problemSize % MaxGridDim.x * MaxGridDim.y * blockSize;
    }

    if (problemSize > MaxGridDim.x * blockSize) {
        gridDim.y = problemSize / MaxGridDim.x * blockSize;
        problemSize = problemSize % MaxGridDim.x * blockSize;
    }

    gridDim.x = (problemSize + blockSize - 1) / blockSize;*/

    return dim3((problemSize + BLOCK_SIZE - 1) / BLOCK_SIZE);
}

///////////////////////////////////////
/// Morton code part {
static __global__
void computeMortonCodesAndReferenceKernel(gpudb::MortonCode *keys, int *values, gpudb::AABB *aabb, gpudb::AABB globalAABB, uint32_t size) {
    uint thread = getGlobalIdx3DZXY();
    if (thread >= size) {
        return;
    }

    keys[thread] = aabb[thread].getMortonCode(globalAABB);
    values[thread] = thread;
}

void computeMortonCodesAndReference(gpudb::MortonCode *keys, int *values, gpudb::AABB *aabb, gpudb::AABB globalAABB, uint size) {
    dim3 block = dim3(BLOCK_SIZE);
    dim3 grid = gridConfigure(size, block);
    computeMortonCodesAndReferenceKernel<<<grid,block>>>(keys, values, aabb, globalAABB, size);
}

__global__
void initKeys(uint64_t *keys, int *values, gpudb::MortonCode *codes, uint size) {
    uint thread = getGlobalIdx3DZXY();
    if (thread >= size) {
        return;
    }
    keys[thread] = codes[thread].high;
    values[thread] = thread;
}

__global__
void computeDiff(uint64_t *keys, int *array, uint size) {
    uint thread = getGlobalIdx3DZXY();
    if (thread >= size) {
        return;
    }

    if ((thread + 1) < size && keys[thread] != keys[thread + 1])  {
        array[thread] = 1;
    } else {
        array[thread] = 0;
    }
}


template<bool high> __global__
void writeNewKeys(uint64_t *keys, int *values, gpudb::MortonCode *codes, int *prefixSum, uint size) {
    uint thread = getGlobalIdx3DZXY();
    if (thread >= size) {
        return;
    }
    keys[thread] = 0;
    if (high) {
        keys[thread] = ((uint64_t)prefixSum[thread]) << 32ULL | ((codes[values[thread]].low & 0xFFFFFFFF00000000ULL) >> 32ULL);
    } else {
        keys[thread] = ((uint64_t)prefixSum[thread]) << 32ULL | ((codes[values[thread]].low & 0x00000000FFFFFFFFULL));
    }
}

__global__
void copyKeys(gpudb::MortonCode *new_keys, gpudb::MortonCode *old_keys, int *new_values, int *old_values, uint size) {
    uint thread = getGlobalIdx3DZXY();
    if (thread >= size) {
        return;
    }

    new_keys[thread].bits = old_keys[old_values[thread]].bits;
    new_keys[thread].low = old_keys[old_values[thread]].low;
    new_keys[thread].high = old_keys[old_values[thread]].high;
    new_values[thread] = old_values[thread];
}

bool sortMortonCodes(gpudb::MortonCode *keys, int *values, uint size) {
    uint64_t *cub_keys[2];
    int *cub_values[2];
    dim3 block = dim3(BLOCK_SIZE);
    dim3 grid = gridConfigure(size, block);
    gpudb::GpuStackAllocator::getInstance().pushPosition();
    int switcher = 0;
    for (int i = 0; i < 2; i++) {
        cub_keys[i] = gpudb::GpuStackAllocator::getInstance().alloc<uint64_t>(size);
        cub_values[i]  = gpudb::GpuStackAllocator::getInstance().alloc<int>(size);
    }

    size_t cub_tmp_memory_size = 0;
    size_t cub_tmp_memory_size2 = 0;
    hipcub::DeviceRadixSort::SortPairs(nullptr, cub_tmp_memory_size, cub_keys[0], cub_keys[0], cub_keys[0], cub_keys[0], size);
    hipcub::DeviceScan::ExclusiveSum(nullptr, cub_tmp_memory_size2, cub_values[0], cub_values[0], size);
    cub_tmp_memory_size = std::max(cub_tmp_memory_size, cub_tmp_memory_size2);
    void *cub_tmp_memory  = (void *)gpudb::GpuStackAllocator::getInstance().alloc<uint8_t>(cub_tmp_memory_size);

    if (cub_keys[0] == nullptr || cub_values[0] == nullptr ||
        cub_keys[1] == nullptr || cub_values[1] == nullptr ||
        cub_tmp_memory == nullptr) {
        gpudb::GpuStackAllocator::getInstance().popPosition();
        return false;
    }
    // закончили с выделением памяти, фух

    // первая сортировка
    initKeys<<<grid, block>>>(cub_keys[switcher], cub_values[switcher], keys, size);
    hipcub::DeviceRadixSort::SortPairs(cub_tmp_memory, cub_tmp_memory_size,
                                    cub_keys[switcher], cub_keys[1 - switcher],
                                    cub_values[switcher], cub_values[1 - switcher], size);
    switcher = 1 - switcher;
    // теперь нам надо ещё 2. Чтобы сохранить порядок, посчитаем префиксную сумму
    // и в качестве ключа будем использовать
    // prefixsum << 32 | keypart

    // отсортированное в cub_keys/cub_values[switcher], эти два массива теперь не нужны

    int *arrayPrefixSum = reinterpret_cast<int*>(cub_keys[1 - switcher]);
    int *array = reinterpret_cast<int*>(cub_values[1 - switcher]);
    computeDiff<<<grid, block>>>(cub_keys[switcher], array, size);
    hipcub::DeviceScan::ExclusiveSum(cub_tmp_memory, cub_tmp_memory_size, array, arrayPrefixSum, size);
    writeNewKeys<true><<<grid, block>>>(cub_keys[switcher], cub_values[switcher], keys, arrayPrefixSum, size);
    hipcub::DeviceRadixSort::SortPairs(cub_tmp_memory, cub_tmp_memory_size,
                                    cub_keys[switcher], cub_keys[1 - switcher],
                                    cub_values[switcher], cub_values[1 - switcher], size);

    switcher = 1 - switcher;
    arrayPrefixSum = reinterpret_cast<int*>(cub_keys[1 - switcher]);
    array = reinterpret_cast<int*>(cub_values[1 - switcher]);
    computeDiff<<<grid, block>>>(cub_keys[switcher], array, size);
    hipcub::DeviceScan::ExclusiveSum(cub_tmp_memory, cub_tmp_memory_size, array, arrayPrefixSum, size);
    writeNewKeys<false><<<grid, block>>>(cub_keys[switcher], cub_values[switcher], keys, arrayPrefixSum, size);
    hipcub::DeviceRadixSort::SortPairs(cub_tmp_memory, cub_tmp_memory_size,
                                    cub_keys[switcher], cub_keys[1 - switcher],
                                    cub_values[switcher], cub_values[1 - switcher], size);

    // памяти от cub_keys[0] до cub_values[0] должно хватить, чтобы вместить копию мортон кодов...
  //  hipMemcpy(cub_values[switcher], cub_values[1 - switcher], sizeof(int) * size, hipMemcpyDeviceToDevice);
    gpudb::MortonCode *old = reinterpret_cast<gpudb::MortonCode *> (cub_keys[0]);
    hipMemcpy(old, keys, sizeof(gpudb::MortonCode) * size, hipMemcpyDeviceToDevice);
    copyKeys<<<grid, block>>>(keys, old, values, cub_values[1], size);
    hipDeviceSynchronize();
    gpudb::GpuStackAllocator::getInstance().popPosition();
    return true;
}
/// } Morton code part
///////////////////////////////////////
/// Global AABB {
template<char comp, bool min>
__global__
void copyAABBComponent(float *dst, gpudb::AABB *aabb, uint32_t size) {
    uint thread = getGlobalIdx3DZXY();
    if (thread >= size) {
        return;
    }

    switch(comp) {
        case 'x':
        {
            if (min) {
                dst[thread] = AABBmin(aabb[thread].x);
            } else {
                dst[thread] = AABBmax(aabb[thread].x);
            }
        }
        break;
        case 'y':
        {
            if (min) {
                dst[thread] = AABBmin(aabb[thread].y);
            } else {
                dst[thread] = AABBmax(aabb[thread].y);
            }
        }
        break;
        case 'z':
        {
            if (min) {
                dst[thread] = AABBmin(aabb[thread].z);
            } else {
                dst[thread] = AABBmax(aabb[thread].z);
            }
        }
        break;
        case 'w':
        {
            if (min) {
                dst[thread] = AABBmin(aabb[thread].w);
            } else {
                dst[thread] = AABBmax(aabb[thread].w);
            }
        }
        break;
    }
}

bool computeGlobalAABB(gpudb::AABB *aabb, uint32_t size, gpudb::AABB &result) {
    gpudb::GpuStackAllocator::getInstance().pushPosition();
    StackAllocator::getInstance().pushPosition();
    do {
        float *array = gpudb::GpuStackAllocator::getInstance().alloc<float>(size);
        float *minmax = gpudb::GpuStackAllocator::getInstance().alloc<float>(8);
        float *cpuMinMax = StackAllocator::getInstance().alloc<float>(8);
        size_t cub_tmp_memory_size = 0;


        hipcub::DeviceReduce::Min(nullptr, cub_tmp_memory_size, array, minmax, size);
        uint8_t *cub_tmp_memory = gpudb::GpuStackAllocator::getInstance().alloc<uint8_t>(cub_tmp_memory_size);

        if (array == nullptr || minmax == nullptr || cub_tmp_memory == nullptr || cpuMinMax == nullptr) { break; }

        dim3 block = dim3(BLOCK_SIZE);
        dim3 grid = gridConfigure(size, block);
        copyAABBComponent<'x', true> <<<grid, block>>> (array, aabb, size);
        hipcub::DeviceReduce::Min(cub_tmp_memory, cub_tmp_memory_size, array, minmax + 0, size);
        copyAABBComponent<'y', true> <<<grid, block>>> (array, aabb, size);
        hipcub::DeviceReduce::Min(cub_tmp_memory, cub_tmp_memory_size, array, minmax + 1, size);
        copyAABBComponent<'z', true> <<<grid, block>>> (array, aabb, size);
        hipcub::DeviceReduce::Min(cub_tmp_memory, cub_tmp_memory_size, array, minmax + 2, size);
        copyAABBComponent<'w', true> <<<grid, block>>> (array, aabb, size);
        hipcub::DeviceReduce::Min(cub_tmp_memory, cub_tmp_memory_size, array, minmax + 3, size);

        copyAABBComponent<'x', false> <<<grid, block>>> (array, aabb, size);
        hipcub::DeviceReduce::Max(cub_tmp_memory, cub_tmp_memory_size, array, minmax + 4, size);
        copyAABBComponent<'y', false> <<<grid, block>>> (array, aabb, size);
        hipcub::DeviceReduce::Max(cub_tmp_memory, cub_tmp_memory_size, array, minmax + 5, size);
        copyAABBComponent<'z', false> <<<grid, block>>> (array, aabb, size);
        hipcub::DeviceReduce::Max(cub_tmp_memory, cub_tmp_memory_size, array, minmax + 6, size);
        copyAABBComponent<'w', false> <<<grid, block>>> (array, aabb, size);
        hipcub::DeviceReduce::Max(cub_tmp_memory, cub_tmp_memory_size, array, minmax + 7, size);

        hipMemcpy(cpuMinMax, minmax, sizeof(float) * 8, hipMemcpyDeviceToHost);

        result.x.x = cpuMinMax[0];
        result.y.x = cpuMinMax[1];
        result.z.x = cpuMinMax[2];
        result.w.x = cpuMinMax[3];

        result.x.y = cpuMinMax[4];
        result.y.y = cpuMinMax[5];
        result.z.y = cpuMinMax[6];
        result.w.y = cpuMinMax[7];

        gpudb::GpuStackAllocator::getInstance().popPosition();
        StackAllocator::getInstance().popPosition();
        return true;
    } while(0);

    gLogWrite(LOG_MESSAGE_TYPE::ERROR, "not enough memory");
    gpudb::GpuStackAllocator::getInstance().popPosition();
    StackAllocator::getInstance().popPosition();
    return false;
}
/// } Global AABB
//////////////////////////////////////////
/// Build Tree topology {

struct WorkQueue {
    int *nodeId;
    uint2 *range;
};

void initQueue(WorkQueue &queue, uint32_t size) {
    int nodeId = 0;
    uint2 range;
    range.x = 0;
    range.y = size;
    hipMemcpy(queue.nodeId, &nodeId, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(queue.range, &range, sizeof(uint2), hipMemcpyHostToDevice);
}

#define getLeftBound(p) p.x
#define getRightBound(p) p.y
#define getRangeSize(p) (p.y - p.x)

#define clzll(x) __clzll((x))
#define clzllHost(x) ((x) == 0)? 64 : __builtin_clzll((x))

__global__
void split(gpudb::HLBVH hlbvh, gpudb::MortonCode *keys, uint32_t queueSize, WorkQueue qIn, WorkQueue qOut, uint *counter) {
    uint thread = getGlobalIdx3DZXY();
    if (thread >= queueSize) {
        return;
    }
    uint2 rangeLeft, rangeRight, range;
    range = qIn.range[thread];
    rangeRight = range;
    rangeLeft = range;

    bool isLeaf = true;

    int parent = qIn.nodeId[thread];
    if (getRangeSize(range) > 1) {
        isLeaf = false;
        gpudb::MortonCode keyA = keys[getLeftBound(range)];
        gpudb::MortonCode keyB = keys[getRightBound(range) - 1];
        uint64_t ha = 64;
        uint64_t mask;

        bool high = false;
        if (keyA.high != keyB.high) {
            ha = clzll(keyA.high ^ keyB.high);
            high = true;
        } else if (keyA.low != keyB.low) {
            ha = clzll(keyA.low ^ keyB.low);
        }

        if (ha == 64) {
            uint mid = getLeftBound(range) + (getRightBound(range) - getLeftBound(range)) / 2;
            getRightBound(rangeLeft) = getLeftBound(rangeRight) = mid;
        } else {
            mask = 1ULL << (64 - ha - 1);
            uint left, right;
            left = getLeftBound(range);
            right = getRightBound(range);
            bool test;
            uint mid;

            while (left < right) {
                mid = left + (right - left) / 2;
                if (high) {
                    test = (keys[mid].high & mask) > 0;
                } else {
                    test = (keys[mid].low & mask) > 0;
                }
                /* key[mid] > key[left] */
                if (test) {
                    right = mid;
                } else {
                    left = mid + 1;
                }
            }

            getRightBound(rangeLeft) = left;
            getLeftBound(rangeRight) = left;
        }
    }

    hlbvh.ranges[parent] = range;
    if (isLeaf) {
        hlbvh.links[parent] = 0xFFFFFFFFU; // лист
    } else {
        uint offset = atomicAdd(counter, 2);
        uint left = hlbvh.numNodes + offset;

        hlbvh.links[parent] = left;

        qOut.nodeId[offset] = left;
        qOut.nodeId[offset + 1] = left + 1;
        qOut.range[offset] = rangeLeft;
        qOut.range[offset + 1] = rangeRight;
    }
}

bool buildTreeStructure(gpudb::HLBVH &hlbvh, gpudb::MortonCode *keys, int *values, uint32_t size) {
    gpudb::GpuStackAllocator::getInstance().pushPosition();
    StackAllocator::getInstance().pushPosition();

    do {
        WorkQueue work[2];
        work[0].nodeId = gpudb::GpuStackAllocator::getInstance().alloc<int>(size);
        work[0].range = gpudb::GpuStackAllocator::getInstance().alloc<uint2>(size);
        work[1].nodeId = gpudb::GpuStackAllocator::getInstance().alloc<int>(size);
        work[1].range = gpudb::GpuStackAllocator::getInstance().alloc<uint2>(size);

        uint* counter = gpudb::gpuAllocator::getInstance().alloc<uint>(400);
        if (work[0].nodeId == nullptr || work[0].range == nullptr
            || work[1].nodeId == nullptr || work[1].range == nullptr
            || counter == nullptr)
        {
            break;
        }

        int switcher = 0;
        hipMemset(counter, 0, sizeof(uint) * 400);
        initQueue(work[switcher], size);
        uint queueSize = 1;
        dim3 block = dim3(BLOCK_SIZE);
        dim3 grid;
        while(queueSize > 0) {
            grid = gridConfigure(queueSize, block);
            split<<<grid, block>>>(hlbvh, keys, queueSize, work[switcher], work[1 - switcher], counter);
            hipMemcpy(&queueSize, counter, sizeof(uint), hipMemcpyDeviceToHost);
            hlbvh.numNodes += queueSize;
            switcher = 1 - switcher;
            counter++;
        }
        gpudb::GpuStackAllocator::getInstance().popPosition();
        StackAllocator::getInstance().popPosition();
        return true;
    } while(0);

    gLogWrite(LOG_MESSAGE_TYPE::ERROR, "not enough memory");
    gpudb::GpuStackAllocator::getInstance().popPosition();
    StackAllocator::getInstance().popPosition();
    return false;
}
/// } Build Tree topology
/////////////////////////////////////////////
void test(gpudb::MortonCode *keys, uint32_t bitshift, uint2 range, uint2 &rangeLeft, uint2 &rangeRight) {
    rangeRight = rangeLeft = range;

    gpudb::MortonCode keyA = keys[getLeftBound(range)];
    gpudb::MortonCode keyB = keys[getRightBound(range) - 1];
    uint64_t ha = 64;

    uint64_t hb = 64;
    uint64_t mask;
    bool high = false;
    if (bitshift > 64) {
        if (keyA.high != keyB.high) {
            mask = (1ULL << (bitshift - 64)) - 1;
            ha = keyA.high & mask;
            hb = keyB.high & mask;
            ha = clzllHost(ha ^ hb);
            high = true;
        } else {
            ha = clzllHost(keyA.low ^ keyB.low);
        }
    } else {
        mask = (1ULL << (bitshift)) - 1;
        ha = keyA.low & mask;
        hb = keyB.low & mask;
        ha = clzllHost(ha ^ hb);
    }
/*
     bool high = false;
     if (keyA.high != keyB.high) {
         ha = clzllHost(keyA.high ^ keyB.high);
         high = true;
     } else {
         ha = clzllHost(keyA.low ^ keyB.low);
     }*/

    if (ha == 64) {
        uint mid = getLeftBound(range) + (getRightBound(range) - getLeftBound(range)) / 2;
        getRightBound(rangeLeft) = getLeftBound(rangeRight) = mid;
    } else {
        mask = 1ULL << (64 - ha - 1);
        uint left, right;
        left = getLeftBound(range);
        right = getRightBound(range);
        while (left < right) {
            uint mid = left + (right - left) / 2;
            bool test;
            if (high) {
                test = (keys[mid].high & mask) > 0;
            } else {
                test = (keys[mid].low & mask) > 0;
            }
            /* key[mid] > key[left] */
            if (test) {
                right = mid;
            } else {
                left = mid + 1;
            }
        }
        getRightBound(rangeLeft) = left;
        getLeftBound(rangeRight) = left;
    }
    printf("%s \n", keys[rangeLeft.x].toString().c_str());
    printf("%s \n", keys[rangeLeft.y - 1].toString().c_str());
    printf("%s \n", keys[rangeRight.x].toString().c_str());
    printf("%s \n", keys[rangeRight.y - 1].toString().c_str());
}

bool gpudb::HLBVH::build(AABB *aabb, uint32_t size) {
    if (size == 0) { return false; }
    gpudb::GpuStackAllocator &gpuStackAlloc = gpudb::GpuStackAllocator::getInstance();
    if (alloc(size) == false) {
        return false;
    }

    do {
        AABB globalAABB;
        if (!computeGlobalAABB(aabb, size, globalAABB)) { break; }

        int *values = gpuStackAlloc.alloc<int>(size);
        MortonCode *keys = gpuStackAlloc.alloc<MortonCode>(size);
        if (keys == nullptr || values == nullptr) { break; }
        computeMortonCodesAndReference(keys, values, aabb, globalAABB, size);
        MortonCode *cpuKeys = StackAllocator::getInstance().alloc<MortonCode>(size);
        int *cpuValues = StackAllocator::getInstance().alloc<int>(size);
        MortonCode *cpuKeys2 = StackAllocator::getInstance().alloc<MortonCode>(size);
        int *cpuValues2 = StackAllocator::getInstance().alloc<int>(size);

        hipMemcpy(cpuKeys2, keys, sizeof(MortonCode) * size, hipMemcpyDeviceToHost);
        hipMemcpy(cpuValues2, values, sizeof(int) * size, hipMemcpyDeviceToHost);

        hipMemcpy(cpuKeys, keys, sizeof(MortonCode) * size, hipMemcpyDeviceToHost);
        hipMemcpy(cpuValues, values, sizeof(int) * size, hipMemcpyDeviceToHost);
        //for (int i = 0; i < size; i++) {
        //    printf("{ (%s), %d }\n", cpuKeys[i].toString().c_str(), cpuValues[i]);
       // }

        if (!sortMortonCodes(keys, values, size)) { break; }
        if (!buildTreeStructure(*this, keys, values, size)) { break; }


        hipMemcpy(cpuKeys, keys, sizeof(MortonCode) * size, hipMemcpyDeviceToHost);
        hipMemcpy(cpuValues, values, sizeof(int) * size, hipMemcpyDeviceToHost);
        for (int i = 0; i < size; i++) {
            //printf("{ (%s), %d }\n", cpuKeys[i].toString().c_str(), cpuValues[i]);
            cpuKeys2[i].bits = cpuValues2[i];
        }
        //printf("\n\n\n");
        std::sort(cpuKeys2, cpuKeys2 + size);

        /*for (int i = 0; i < size; i++) {
            if (cpuValues[i] != cpuKeys2[i].bits) {
                printf("{ %d }\n", cpuValues[i] == cpuKeys2[i].bits);
            }
        }*/
        /*uint2 range;
        range.x = 0;
        range.y = size;
        uint2 rangeL;
        uint2 rangeR;
        test(cpuKeys, 96, range, rangeL, rangeR);
        printf("{%d %d} {%d %d}\n", rangeL.x, rangeL.y, rangeR.x, rangeR.y);
        test(cpuKeys, 95, rangeR, rangeL, rangeR);
        printf("{%d %d} {%d %d}\n", rangeL.x, rangeL.y, rangeR.x, rangeR.y);
        test(cpuKeys, 94, rangeR, rangeL, rangeR);
        printf("{%d %d} {%d %d}\n", rangeL.x, rangeL.y, rangeR.x, rangeR.y);
        test(cpuKeys, 93, rangeR, rangeL, rangeR);
        printf("{%d %d} {%d %d}\n", rangeL.x, rangeL.y, rangeR.x, rangeR.y);
        test(cpuKeys, 93, rangeR, rangeL, rangeR);
        printf("{%d %d} {%d %d}\n", rangeL.x, rangeL.y, rangeR.x, rangeR.y);
        test(cpuKeys, 93, rangeR, rangeL, rangeR);
        printf("{%d %d} {%d %d}\n", rangeL.x, rangeL.y, rangeR.x, rangeR.y);*/
        gLogWrite(LOG_MESSAGE_TYPE::DEBUG, "end hlbvh build");
        gpuStackAlloc.popPosition();
        return true;
    } while(0);
    gLogWrite(LOG_MESSAGE_TYPE::ERROR, "not enough memory");
    gpuStackAlloc.popPosition();
    return false;
}
