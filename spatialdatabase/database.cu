#include "hip/hip_runtime.h"
#include "database.h"
#include <cub/hipcub/hipcub.hpp>
#include "make_unique.h"

__device__ void
gpuRowGpuToGpu(gpudb::GpuRow * dst, gpudb::GpuRow const * src, uint64_t const memsize) {
    memcpy(dst, src, memsize);
    dst->spatialPart.key = newAddress(dst->spatialPart.key, src, dst);
    switch (dst->spatialPart.type) {
        case SpatialType::POINT:
        break;
        case SpatialType::LINE:
        {
            gpudb::GpuLine *line = ((gpudb::GpuLine*)(dst->spatialPart.key));
            line->points = newAddress(line->points, src, dst);
        }
        break;
        case SpatialType::POLYGON:
        {
            gpudb::GpuPolygon *polygon = ((gpudb::GpuPolygon*)(dst->spatialPart.key));
            polygon->points = newAddress(polygon->points, src, dst);
        }
        break;
    }

    dst->value = newAddress(dst->value, src, dst);

    for (uint i = 0; i < dst->valueSize; i++) {
        dst->value[i].value = newAddress(dst->value[i].value, src, dst);
    }
}

__global__
void gpuRowsCopy(gpudb::GpuRow **dst, gpudb::GpuRow * const *src, uint64_t *sizes, uint count) {
    uint idx = getGlobalIdx3DZXY();
    if (idx >= count) { return; }

    gpuRowGpuToGpu(dst[idx], src[idx], sizes[idx]);
}

__global__ void
gpuRowsCopyOnlySelected(gpudb::GpuRow **dst, gpudb::GpuRow * const *src, uint *selectors,
                        uint64_t *sizes, uint count) {
    uint idx = getGlobalIdx3DZXY();
    if (idx >= count) { return; }
    gpuRowGpuToGpu(dst[idx], src[selectors[idx]], sizes[idx]);
}

Result<std::unique_ptr<TempTable>, Error<std::string>>
DataBase::copyTempTable(TableDescription const &description, gpudb::GpuTable const *gpuTable) {
    if (gpuTable->rows.size() == 0) {
        return MYERR_STRING("table has no rows");
    }

    std::unique_ptr<TempTable> result = std::make_unique<TempTable>();

    if (result == nullptr) {
        return MYERR_STRING("not enough ram memory");
    }

    result->description = description;
    result->table = new gpudb::GpuTable;

    if (result->table == nullptr) {
        return MYERR_STRING("not enough ram memory");
    }

    result->table->columns.reserve(gpuTable->columns.size());
    result->table->columns = gpuTable->columns;
    memcpy(result->table->name, gpuTable->name, NAME_MAX_LEN * sizeof(char));
    result->table->rows.reserve(gpuTable->rows.size());
    std::vector<gpudb::GpuRow *> hostRows(gpuTable->rows.size());
    auto sizes = gpudb::GpuStackAllocatorAdditions::allocUnique<uint64_t>(hostRows.size());

    if (sizes == nullptr) {
        return MYERR_STRING("not enough gpu stack memory");
    }

    for (size_t i = 0; i < hostRows.size(); i++) {
        hostRows[i] = (gpudb::GpuRow*) gpudb::gpuAllocator::getInstance().alloc<uint8_t>(gpuTable->rowsSize[i]);
        if (hostRows[i] == nullptr) {
            for (size_t j = 0; j < i; j++) {
                gpudb::gpuAllocator::getInstance().free(hostRows[j]);
            }
            return MYERR_STRING("not enough gpu memory");
        }
    }

    hipMemcpy(sizes.get(), gpuTable->rowsSize.data(), sizeof(uint64_t) * hostRows.size(), hipMemcpyHostToDevice);
    result->table->rows = hostRows;
    result->table->rowsSize = gpuTable->rowsSize;

    dim3 grid = gridConfigure(hostRows.size(), BLOCK_SIZE);
    dim3 block = dim3(BLOCK_SIZE);

    gpuRowsCopy<<<grid, block>>>(thrust::raw_pointer_cast(result->table->rows.data()),
                                 thrust::raw_pointer_cast(gpuTable->rows.data()),
                                 sizes.get(),
                                 hostRows.size());
    result->valid = true;
    return Ok(std::move(result));
}

Result<std::unique_ptr<TempTable>, Error<std::string>>
DataBase::selectTable(std::string tableName) {
    auto descriptionIt = tablesType.find(tableName);
    auto tableIt = tables.find(tableName);

    if (descriptionIt == tablesType.end() || tableIt == tables.end()) {
        return MYERR_STRING(string_format("Table with name %s was not found", tableName.c_str()));
    }

    TableDescription &refTableDescription = descriptionIt->second;
    gpudb::GpuTable *pointerGpuTable = tableIt->second;

    return copyTempTable(refTableDescription, pointerGpuTable);
}

__global__
void buildKeysAABB(gpudb::GpuRow** rows, gpudb::AABB *boxes, uint size) {
    uint idx = getGlobalIdx3DZXY();
    if (idx >= size) {
        return;
    }

    rows[idx]->spatialPart.boundingBox(&boxes[idx]);
    AABBmin(boxes[idx].z) = AABBmax(boxes[idx].z) = 0.0f;
    AABBmin(boxes[idx].w) = AABBmax(boxes[idx].w) = 0.0f;
    //rows[idx]->temporalPart.boundingBox(&boxes[idx]);
}

__device__
AABBRelation boxIntersection2D(float4 aMin, float4 aMax, float4 bMin, float4 bMax) {
    bool a1 = aMax.x < bMin.x;
    bool a2 = aMin.x > bMax.x;

    bool a3 = aMax.y < bMin.y;
    bool a4 = aMin.y > bMax.y;

   /* bool a5 = aMax.z < bMin.z;
    bool a6 = aMin.z > bMax.z;

    bool a7 = aMax.w < bMin.w;
    bool a8 = aMin.w > bMax.w;*/

    if (a1 || a2 || a3 || a4 /*&& a5 &&a6 && a7 && a8*/) {
        return AABBRelation::DISJOINT;
    }

    if (aMin.x <= bMin.x && bMin.x <= aMax.x &&
        aMin.x <= bMax.x && bMax.x <= aMax.x &&
        aMin.y <= bMin.y && bMin.y <= aMax.y &&
        aMin.y <= bMax.y && bMax.y <= aMax.y /*&&
        aMin.z <= bMin.z && bMin.z <= aMax.z &&
        aMin.z <= bMax.z && bMax.z <= aMax.z &&
        aMin.w <= bMin.w && bMin.w <= aMax.w &&
        aMin.w <= bMax.w && bMax.w <= aMax.w*/) {
        return AABBRelation::BINSIDEA;
    }

    return AABBRelation::OVERLAP;
}


__device__
void computeBoundingBoxLineBuffer(gpudb::GpuLine *line, float radius, float2 &min2, float2 &max2) {
    min2 = make_float2(INFINITY, INFINITY);
    max2 = make_float2(-INFINITY, -INFINITY);

    float2 s = line->points[0];
    float2 e = line->points[1]; // предпологаем что линия не из одной точки

    float2 dir = e - s;
    dir = norma(dir);
    float2 n = make_float2(dir.y, -dir.x);
    float2 resizedN = n * radius;
    float2 resizedDir = dir * radius;

    float2 p1 = s - resizedDir;

    min2 = fmin(p1, min2);
    max2 = fmax(p1, max2);

    float2 p1_1 = p1 + resizedN;
    float2 p1_2 = p1 - resizedN;

    min2 = fmin(p1_1, min2);
    min2 = fmin(p1_2, min2);
    max2 = fmax(p1_1, max2);
    max2 = fmax(p1_2, max2);

    for (uint i = 0; i < line->size - 1; i++) {
        s = line->points[i];
        e = line->points[i + 1];
        dir = e - s;
        dir = norma(dir);
        n = make_float2(dir.y, -dir.x);
        resizedN = n * radius;
        resizedDir = dir * radius;
        p1 = e + resizedDir;
        p1_1 = p1 + resizedN;
        p1_2 = p1 - resizedN;
        min2 = fmin(p1, min2);
        max2 = fmax(p1, max2);
        min2 = fmin(p1_1, min2);
        min2 = fmin(p1_2, min2);
        max2 = fmax(p1_1, max2);
        max2 = fmax(p1_2, max2);
    }
}

__device__
void boxInsideBoxSubKernel(float4 aabbMin, float4 aabbMax, uint &numBoxBinside, gpudb::HLBVH &bvh, uint *stack, uint stackSize) {
    GpuStack<uint> st(stack, stackSize);
    st.push(0);
    uint sum = 0;
    while(!st.empty()) {
        uint pos = st.top(); st.pop();
        float4 aMax = bvh.aabbMax[pos];
        float4 aMin = bvh.aabbMin[pos];
        float4 bMax = bvh.aabbMax[pos + 1];
        float4 bMin = bvh.aabbMin[pos + 1];
        int link1 = bvh.links[pos];
        int link2 = bvh.links[pos + 1];
        AABBRelation r1 = boxIntersection2D(aabbMin, aabbMax, aMin, aMax);
        AABBRelation r2 = boxIntersection2D(aabbMin, aabbMax, bMin, bMax);

        if (r1 == AABBRelation::OVERLAP && link1 != LEAF) {
            st.push(link1);
        }

        if (r2 == AABBRelation::OVERLAP && link2 != LEAF) {
            st.push(link2);
        }

        if (r1 == AABBRelation::BINSIDEA || (r1 == AABBRelation::OVERLAP && link1 == LEAF)) {
            sum += getRangeSize(bvh.ranges[pos]);
        }

        if (r2 == AABBRelation::BINSIDEA || (r2 == AABBRelation::OVERLAP && link2 == LEAF)) {
            sum += getRangeSize(bvh.ranges[pos + 1]);
        }
    }
    numBoxBinside = sum;
}

__device__
void boxInsideBoxSubKernel(uint idx, float4 aabbMin, float4 aabbMax, uint numBoxBInside, uint offsets, uint *testedBoxANum,
                           uint *testedBoxBNum, gpudb::HLBVH &bvh, uint *stack, uint stackSize) {
    GpuStack<uint> st(stack, stackSize);
    uint num = numBoxBInside;
    uint pointCounter = 0;
    st.push(0);
    while(!st.empty() && pointCounter < num) {
        uint pos = st.top(); st.pop();
        float4 aMax = bvh.aabbMax[pos];
        float4 aMin = bvh.aabbMin[pos];
        float4 bMax = bvh.aabbMax[pos + 1];
        float4 bMin = bvh.aabbMin[pos + 1];
        int link1 = bvh.links[pos];
        int link2 = bvh.links[pos + 1];
        AABBRelation r1 = boxIntersection2D(aabbMin, aabbMax, aMin, aMax);
        AABBRelation r2 = boxIntersection2D(aabbMin, aabbMax, bMin, bMax);

        if (r1 == AABBRelation::OVERLAP && link1 != LEAF) {
            st.push(link1);
        }

        if (r2 == AABBRelation::OVERLAP && link2 != LEAF) {
            st.push(link2);
        }

        if ((r1 == AABBRelation::BINSIDEA) || (r1 == AABBRelation::OVERLAP && link1 == LEAF)) {
            for (uint i = getLeftBound(bvh.ranges[pos]); i < getRightBound(bvh.ranges[pos]); i++) {
                testedBoxBNum[offsets + pointCounter] = bvh.references[i];
                testedBoxANum[offsets + pointCounter] = idx;
                pointCounter++;
            }
        }

        if ((r2 == AABBRelation::BINSIDEA) || (r2 == AABBRelation::OVERLAP && link2 == LEAF)) {
            for (uint i = getLeftBound(bvh.ranges[pos + 1]); i < getRightBound(bvh.ranges[pos + 1]); i++) {
                testedBoxBNum[offsets + pointCounter] = bvh.references[i];
                testedBoxANum[offsets + pointCounter] = idx;
                pointCounter++;
            }
        }
    }
}

__global__
void boxInsideBoxKernel(gpudb::GpuRow **rowsBoxA, uint *numBoxBinside, gpudb::HLBVH bvh, uint *stack, uint stackSize, uint workSize) {
    uint idx = getGlobalIdx3DZXY();
    if (idx >= workSize) {
        return;
    }
    gpudb::AABB box;
    rowsBoxA[idx]->spatialPart.boundingBox(&box);
    rowsBoxA[idx]->temporalPart.boundingBox(&box);

    float4 min = make_float4(box.x.x, box.y.x, box.z.x, box.w.x);
    float4 max = make_float4(box.x.y, box.y.y, box.z.y, box.w.y);

    boxInsideBoxSubKernel(min, max, numBoxBinside[idx], bvh, &stack[idx * stackSize], stackSize);
}

__global__
void boxInsideBoxKernel2(gpudb::GpuRow **rowsBoxA, uint *numBoxBInside, uint *offsets, uint *testedBoxANum, uint *testedBoxBNum,
                         gpudb::HLBVH bvh, uint *stack, uint stackSize, uint workSize) {
    uint idx = getGlobalIdx3DZXY();
    if (idx >= workSize) {
        return;
    }
    gpudb::AABB box;
    rowsBoxA[idx]->spatialPart.boundingBox(&box);
    rowsBoxA[idx]->temporalPart.boundingBox(&box);

    float4 min = make_float4(box.x.x, box.y.x, box.z.x, box.w.x);
    float4 max = make_float4(box.x.y, box.y.y, box.z.y, box.w.y);

    boxInsideBoxSubKernel(idx, min, max, numBoxBInside[idx], offsets[idx], testedBoxANum, testedBoxBNum, bvh, &stack[idx * stackSize], stackSize);
}


__global__
void boxInsideBoxLineKernel(gpudb::GpuRow **rowsBoxA, uint *numBoxBinside, gpudb::HLBVH bvh, uint *stack, uint stackSize, uint workSize, float radius) {
    uint idx = getGlobalIdx3DZXY();
    if (idx >= workSize) {
        return;
    }
    gpudb::AABB box;
    gpudb::GpuLine *line = (gpudb::GpuLine *)rowsBoxA[idx]->spatialPart.key;
    float2 min2, max2;

    computeBoundingBoxLineBuffer(line, radius, min2, max2);
    rowsBoxA[idx]->temporalPart.boundingBox(&box);

    float4 min = make_float4(min2.x, min2.y, box.z.x, box.w.x);
    float4 max = make_float4(max2.x, max2.y, box.z.y, box.w.y);

    boxInsideBoxSubKernel(min, max, numBoxBinside[idx], bvh, &stack[idx * stackSize], stackSize);
}

__global__
void boxInsideBoxLineKernel2(gpudb::GpuRow **rowsBoxA, uint *numBoxBInside, uint *offsets, uint *testedBoxANum, uint *testedBoxBNum,
                             gpudb::HLBVH bvh, uint *stack, uint stackSize, uint workSize, float radius) {
    uint idx = getGlobalIdx3DZXY();
    if (idx >= workSize) {
        return;
    }
    gpudb::AABB box;
    gpudb::GpuLine *line = (gpudb::GpuLine *)rowsBoxA[idx]->spatialPart.key;
    float2 min2, max2;

    computeBoundingBoxLineBuffer(line, radius, min2, max2);
    rowsBoxA[idx]->temporalPart.boundingBox(&box);

    float4 min = make_float4(min2.x, min2.y, box.z.x, box.w.x);
    float4 max = make_float4(max2.x, max2.y, box.z.y, box.w.y);

    boxInsideBoxSubKernel(idx, min, max, numBoxBInside[idx], offsets[idx], testedBoxANum, testedBoxBNum, bvh, &stack[idx * stackSize], stackSize);
}


__device__
float isLeft(float2 s, float2 e, float2 p) {
    return (e.x - s.x) * (p.y - s.y) - (p.x - s.x) * (e.y - s.y);
}

__global__
void pointInsidePolygonKernel(gpudb::GpuRow **polygons, uint *numPoints, uint *testedPolygonNum, uint *testedPointNum,
                              uint *testResult, gpudb::GpuRow **points, uint sizeWork) {
    uint idx = getGlobalIdx3DZXY();
    if (idx >= sizeWork) {
        return;
    }

    gpudb::GpuPolygon *polygon = (gpudb::GpuPolygon*)polygons[testedPolygonNum[idx]]->spatialPart.key;
    gpudb::GpuPoint *point = (gpudb::GpuPoint*)points[testedPointNum[idx]]->spatialPart.key;
    float2 p = point->p;
    int wn = 0;
    for (uint i = 0; i < polygon->size; i++) {
        float2 s = polygon->points[i];
        float2 e = polygon->points[((i + 1) < polygon->size)? i + 1 : 0];

        float left = isLeft(s, e, p);
        if (s.y <= p.y && e.y > p.y && left > 0) {
            wn++;
        } else {
            if (s.y > p.y && e.y < p.y && left < 0) {
                wn--;
            }
        }
    }
    testResult[idx] = wn != 0;
    /*if (wn != 0) {
        printf("{%f %f}\n", p.x, p.y);
    }*/
}

__device__
bool insideRect(float2 A, float2 B, float2 C, float2 D, float2 point) {
    float2 dir = B - A;
    float2 dirToP = point - A;
    float dot1 = dot(dir, dirToP);

    float2 dir2 = C - B;
    float2 dirToP2 = point - B;
    float dot2 = dot(dir2, dirToP2);

    float2 dir3 = D - C;
    float2 dirToP3 = point - C;
    float dot3 = dot(dir3, dirToP3);

    float2 dir4 = A - D;
    float2 dirToP4 = point - D;
    float dot4 = dot(dir4, dirToP4);

    return ((dot1 > 0) && (dot2 > 0) && (dot3 > 0) && (dot4 > 0)) ||
           ((dot1 < 0) && (dot2 < 0) && (dot3 < 0) && (dot4 < 0));
}

__global__
void pointInsideLineBufferKernel(gpudb::GpuRow **lines, uint *numPoints, uint *testedLineNum, uint *testedPointNum, uint *testResult,
                                 gpudb::GpuRow **points, uint sizeWork, float radius) {
    uint idx = getGlobalIdx3DZXY();
    if (idx >= sizeWork) {
        return;
    }

    gpudb::GpuLine *line = (gpudb::GpuLine *)lines[testedLineNum[idx]]->spatialPart.key;
    gpudb::GpuPoint *point = (gpudb::GpuPoint *)points[testedPointNum[idx]]->spatialPart.key;
    float2 p = point->p;
    float2 s = line->points[0];
    float2 e;

    bool result = (lenSqr(s, p) < radius * radius);
    for (uint i = 0; i < line->size - 1; i++) {
        s = line->points[i];
        e = line->points[i + 1];
        result = result || (lenSqr(e, p) < radius * radius);

        float2 dir = e - s;
        dir = norma(dir);
        float2 n = make_float2(dir.y, -dir.x);

        float2 A, B, C, D;
        float2 resizedN = n * radius;
        A = resizedN + s;
        B = resizedN + e;
        C = e - resizedN;
        D = s - resizedN;
        // это даёт ориентацию либо по часовой, либо против часовой стрелки
        result = result || insideRect(A, B, C, D, p);
        if (result) { break; }
    }

    /*if (result) {
        printf("%d %f %f\n", testedPointNum[idx], p.x, p.y);
    }*/

    testResult[idx] = result;
}

__global__
void computeResultSize(uint *result, uint *resultPrefixSum, uint *testedPrefixSum, uint workSize) {
    uint idx = getGlobalIdx3DZXY();
    if (idx >= workSize) {
        return;
    }

    uint offset = resultPrefixSum[idx + 1]; // сумма a0 + ... + a_idx
    result[idx] = testedPrefixSum[offset];
}

__global__
void computeResultSize2(uint *result, uint workSize) {
    uint idx = getGlobalIdx3DZXY();
    if (idx >= workSize) {
        return;
    }

    uint sub = 0;
    if (idx > 0) {
        sub = result[idx - 1];
    }

    result[idx] = result[idx] - sub;
}


__global__
void selector(uint *dsc, uint *pointNum, uint *tests, uint *offset, uint workSize) {
    uint idx = getGlobalIdx3DZXY();
    if (idx >= workSize) {
        return;
    }

    if (tests[idx]) {
        dsc[offset[idx]] = pointNum[idx];
    }
}

Result<std::unique_ptr<TempTable>, Error<std::string>>
DataBase::resultToTempTable2(std::unique_ptr<TempTable> const &sourceA, std::unique_ptr<TempTable> &sourceB,
                             std::string nameForNewTempTebles,
                             TempTable **newTempTables, std::string nameForResultTempTable) {
    RAII_GC<gpudb::GpuTable> gc;
    gpudb::GpuTable *resultTable =  new gpudb::GpuTable();

    if (resultTable == nullptr) {
        return MYERR_STRING("not enough ram memory");
    }
    gc.registrCPU(resultTable);

    gpudb::GpuColumnAttribute atr;
    std::snprintf(atr.name, NAME_MAX_LEN, nameForNewTempTebles.c_str());
    atr.type = Type::SET;

    AttributeDescription desc;
    desc.name.resize(NAME_MAX_LEN);
    std::snprintf(&desc.name[0], NAME_MAX_LEN, nameForNewTempTebles.c_str());
    desc.type = Type::SET;

    resultTable->bvh.builded = false;
    std::snprintf(resultTable->name, NAME_MAX_LEN, nameForResultTempTable.c_str());
    resultTable->rows.reserve(sourceA->table->rows.size());
    resultTable->rowsSize.resize(sourceA->table->rows.size());
    resultTable->columns.reserve(sourceA->table->columns.size() + 1);
    resultTable->columns = sourceA->table->columns;
    resultTable->columns.push_back(atr);

    TableDescription tdescription;
    tdescription = sourceA->description;
    tdescription.columnDescription.push_back(desc);
    tdescription.name = nameForResultTempTable;

    auto resultRows = StackAllocatorAdditions::allocUnique<uint8_t*>(sourceA->table->rows.size());
    if (resultRows == nullptr) {
        return MYERR_STRING("not enough cpu memory");
    }

    std::vector<gpudb::GpuRow*> hostRowsResult;
    hostRowsResult.resize(sourceA->table->rows.size());

    uint64_t maxSize = 0;
    for (uint i = 0; i < sourceA->table->rows.size(); i++) {
        uint64_t memsize = sourceA->table->rowsSize[i] + sizeof(gpudb::Value) + typeSize(Type::SET);
        maxSize = std::max(memsize, maxSize);
        resultTable->rowsSize[i] = memsize;
        resultRows.get()[i] = gpudb::gpuAllocator::getInstance().alloc<uint8_t>(memsize);
        hostRowsResult[i] = ((gpudb::GpuRow*)(resultRows.get()[i]));
        if (resultRows.get()[i] == nullptr) {
            return MYERR_STRING("not enough gpu memory");
        }

        gc.registrGPU(resultRows.get()[i]);
    }

    auto cpuRow = StackAllocatorAdditions::allocUnique<uint8_t>(maxSize);

    if (cpuRow == nullptr) {
        return MYERR_STRING("not enough cpu memory");
    }

    resultTable->rows = hostRowsResult;

    std::vector<gpudb::GpuRow*> hostRows(sourceA->table->rows.size());
    thrust::copy(sourceA->table->rows.begin(), sourceA->table->rows.end(), hostRows.begin());
    for (uint i = 0; i < sourceA->table->rows.size(); i++) {
        auto aRow = StackAllocatorAdditions::allocUnique<uint8_t>(sourceA->table->rowsSize[i]);
        DataBase::getInstance().loadCPU((gpudb::GpuRow*)(aRow.get()), hostRows[i], sourceA->table->rowsSize[i]);
        gpudb::GpuRow* cpuRowPointer = ((gpudb::GpuRow*)cpuRow.get());
        gpudb::GpuRow* aCpuRowPointer = ((gpudb::GpuRow*)aRow.get());
        uintptr_t cpuRawPointer = ((uintptr_t)cpuRow.get());

        strncpy(cpuRowPointer->spatialPart.name, aCpuRowPointer->spatialPart.name, typeSize(Type::STRING));
        cpuRowPointer->spatialPart.name[typeSize(Type::STRING) - 1] = 0;
        strncpy(cpuRowPointer->temporalPart.name, aCpuRowPointer->temporalPart.name, typeSize(Type::STRING));
        cpuRowPointer->temporalPart.name[typeSize(Type::STRING) - 1] = 0;

        cpuRowPointer->spatialPart.type = aCpuRowPointer->spatialPart.type;
        cpuRowPointer->temporalPart.type = sourceA->description.temporalKeyType;
        cpuRowPointer->valueSize = tdescription.columnDescription.size();
        cpuRowPointer->value = (gpudb::Value*)(cpuRawPointer + sizeof(gpudb::GpuRow));

        uintptr_t memoryValues = cpuRawPointer + sizeof(gpudb::GpuRow) + sizeof(gpudb::Value) * cpuRowPointer->valueSize;
        for (uint j = 0; j < tdescription.columnDescription.size(); j++) {
            cpuRowPointer->value[j].value = (void*)memoryValues;
            if (j < sourceA->description.columnDescription.size()) {
                cpuRowPointer->value[j].isNull = aCpuRowPointer->value[j].isNull;
            } else {
                cpuRowPointer->value[j].isNull = false;
            }

            uint64_t attrSize = typeSize(tdescription.columnDescription[j].type);
            if (j < sourceA->description.columnDescription.size()) {
                memcpy(cpuRowPointer->value[j].value, aCpuRowPointer->value[j].value, attrSize);
            } else {
                gpudb::GpuSet set;
                set.temptable = newTempTables[i];
                set.columns = thrust::raw_pointer_cast(newTempTables[i]->table->columns.data());
                set.rows = thrust::raw_pointer_cast(newTempTables[i]->table->rows.data());
                set.rowsSize = newTempTables[i]->table->rows.size();
                set.columnsSize = newTempTables[i]->table->columns.size();
                memcpy(cpuRowPointer->value[j].value, &set, attrSize);
            }
            memoryValues += attrSize;
        }

        cpuRowPointer->spatialPart.key = (void*)memoryValues;
        switch (tdescription.spatialKeyType) {
            case SpatialType::POINT:
            {
                gpudb::GpuPoint *p1 = ((gpudb::GpuPoint*)(cpuRowPointer->spatialPart.key));
                gpudb::GpuPoint *p2 = ((gpudb::GpuPoint*)(aCpuRowPointer->spatialPart.key));
                p1->p = p2->p;
            }
            break;
            case SpatialType::LINE:
            {
                gpudb::GpuLine *l1 = ((gpudb::GpuLine*)(cpuRowPointer->spatialPart.key));
                gpudb::GpuLine *l2 = ((gpudb::GpuLine*)(aCpuRowPointer->spatialPart.key));
                l1->size = l2->size;
                l1->points = (float2*)(memoryValues + sizeof(gpudb::GpuLine));
                for (int i = 0; i < l2->size; i++) {
                    l1->points[i] = l2->points[i];
                }
            }
            break;
            case SpatialType::POLYGON:
            {
                gpudb::GpuPolygon *p1 = ((gpudb::GpuPolygon*)(cpuRowPointer->spatialPart.key));
                gpudb::GpuPolygon *p2 = ((gpudb::GpuPolygon*)(aCpuRowPointer->spatialPart.key));
                p1->size = p2->size;
                p1->points = (float2*)(memoryValues + sizeof(gpudb::GpuPolygon));
                for (int i = 0; i < p2->size; i++) {
                    p1->points[i] = p2->points[i];
                }
            }
            break;
        }

        cpuRowPointer->temporalPart.transactionTimeCode = aCpuRowPointer->temporalPart.transactionTimeCode;
        cpuRowPointer->temporalPart.validTimeECode = aCpuRowPointer->temporalPart.validTimeECode;
        cpuRowPointer->temporalPart.validTimeSCode = aCpuRowPointer->temporalPart.validTimeSCode;

        DataBase::getInstance().storeGPU((gpudb::GpuRow*)resultRows.get()[i], cpuRowPointer, resultTable->rowsSize[i]);
    }

    auto resultTempTable = std::make_unique<TempTable>();
    sourceB->references.push_back(resultTempTable.get());
    resultTempTable->parents.push_back(sourceB.get());
    resultTempTable->table = resultTable;
    resultTempTable->valid = true;
    resultTempTable->description = tdescription;

    gc.takeCPU();
    gc.takeGPU();
    return Ok(std::move(resultTempTable));
}

Result<std::unique_ptr<TempTable>, Error<std::string>>
DataBase::resultToTempTable1(std::unique_ptr<TempTable> const &a, std::unique_ptr<TempTable> &b,
                             std::string opname, uint *selectedRowsFromB, uint *selectedRowsSize) {
    auto tables = StackAllocatorAdditions::allocUnique<gpudb::GpuTable*>(a->table->rows.size());
    auto newTempTables = StackAllocatorAdditions::allocUnique<TempTable*>(a->table->rows.size());

    if (tables == nullptr || newTempTables == nullptr) {
        return MYERR_STRING("not enough stack memory");
    }

    RAII_GC<gpudb::GpuTable> gcGT;
    RAII_GC<TempTable> gcTT;

    for (uint i = 0; i < a->table->rows.size(); i++) {
        tables.get()[i] = new gpudb::GpuTable;
        newTempTables.get()[i] = new TempTable;
        gcGT.registrCPU(tables.get()[i]);
        gcTT.registrGPU(newTempTables.get()[i]);

        if (tables.get()[i] == nullptr || newTempTables.get()[i] == nullptr) {
            return MYERR_STRING("not enough ram memory");
        }
    }

    thrust::host_vector<gpudb::GpuRow*> rows;
    thrust::host_vector<gpudb::GpuRow*> brows = b->table->rows;

    size_t j = 0;
    for (size_t i = 0; i < a->table->rows.size(); i++) {
        tables.get()[i]->columns.reserve(a->table->columns.size());
        tables.get()[i]->bvh.builded = false;
        tables.get()[i]->columns = a->table->columns;
        memcpy(tables.get()[i]->name, a->table->name, NAME_MAX_LEN * sizeof(char));
        tables.get()[i]->rowReferenses = true;
        if (selectedRowsSize[i] > 0) {
            rows.resize(selectedRowsSize[i]);
            tables.get()[i]->rowsSize.resize(selectedRowsSize[i]);
            tables.get()[i]->rows.reserve(selectedRowsSize[i]);
            for (size_t p = 0; p < selectedRowsSize[i]; p++, j++) {
                tables.get()[i]->rowsSize[p] = b->table->rowsSize[selectedRowsFromB[j]];
                rows[p] = brows[selectedRowsFromB[j]];
            }
            tables.get()[i]->rows = rows;
        }
    }

    for (uint i = 0; i < a->table->rows.size(); i++) {
        newTempTables.get()[i]->description = b->description;
        newTempTables.get()[i]->valid = true;
        newTempTables.get()[i]->table = tables.get()[i];
    }

    std::unique_ptr<TempTable> resultTempTable = TRY(resultToTempTable2(a, b, opname.c_str(), newTempTables.get(), opname.c_str()));
    resultTempTable->insideAllocations.resize(a->table->rows.size());

    for (uint i = 0; i < a->table->rows.size(); i++) {
        resultTempTable->insideAllocations.push_back(newTempTables.get()[i]);
    }

    gcTT.takeCPU();
    gcTT.takeGPU();
    gcGT.takeCPU();
    gcGT.takeGPU();
    return Ok(std::move(resultTempTable));
}

Result<std::unique_ptr<TempTable>, Error<std::string>>
DataBase::linexpointPointsInBufferLine(std::unique_ptr<TempTable> const &a, std::unique_ptr<TempTable> &b, float radius) {
    if (a == nullptr || b == nullptr) {
        return MYERR_STRING("TempTable a or b is nullptr");
    }

    if (!a->isValid() || !b->isValid()) {
        return MYERR_STRING("TempTable a or b is invalid");
    }

    if (a->table == nullptr ||
        b->table == nullptr)
    {
        return MYERR_STRING("a.table  or b.table is nullptr");
    }

    if (a->getSpatialKeyType() != SpatialType::LINE ||
        b->getSpatialKeyType() != SpatialType::POINT)
    {
        return MYERR_STRING("a or b type mismatch");
    }

    if (radius <= 0.0001f) {
        return MYERR_STRING(string_format("radius %d <= 0.0001f", radius));
    }

    if (a->table->rows.size() == 0 || b->table->rows.size() == 0) {
        return MYERR_STRING("TempTable a or b has no rows");
    }

    if (!b->table->bvh.isBuilded()) {
        auto boxes = gpudb::GpuStackAllocatorAdditions::allocUnique<gpudb::AABB> (b->table->rows.size());
        if (boxes == nullptr) {
            return MYERR_STRING("not enough gpu stack memory");
        }

        dim3 block(BLOCK_SIZE);
        dim3 grid(gridConfigure(b->table->rows.size(), block));
        buildKeysAABB<<<grid, block>>>(thrust::raw_pointer_cast(b->table->rows.data()), boxes.get(), b->table->rows.size());
        TRY(b->table->bvh.build(boxes.get(), b->table->rows.size()));
    }

    uint stackSize = b->table->bvh.numBVHLevels * 2 + 1;
    auto stack = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(stackSize * a->table->rows.size());
    auto pointsInsideLineBuffer = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(a->table->rows.size() + 1);
    auto prefixSumPointsInsideLineBuffer = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(a->table->rows.size() + 1);

    uint64_t cub_tmp_memsize = 0;
    hipcub::DeviceScan::ExclusiveSum(nullptr, cub_tmp_memsize, pointsInsideLineBuffer.get(), prefixSumPointsInsideLineBuffer.get(), a->table->rows.size() + 1);
    auto cub_tmp_mem = gpudb::GpuStackAllocatorAdditions::allocUnique<uint8_t>(cub_tmp_memsize);

    auto cpuPointsInsidePolygon = StackAllocatorAdditions::allocUnique<uint>(a->table->rows.size());

    if (pointsInsideLineBuffer == nullptr ||
        stack == nullptr ||
        cpuPointsInsidePolygon == nullptr ||
        cub_tmp_mem == nullptr ||
        prefixSumPointsInsideLineBuffer == nullptr) {
        return MYERR_STRING("not enough stack memory");
    }

    dim3 block(BLOCK_SIZE);
    dim3 grid(gridConfigure(a->table->rows.size(), block));
    boxInsideBoxLineKernel<<<grid, block>>>(thrust::raw_pointer_cast(a->table->rows.data()),
                                                    pointsInsideLineBuffer.get(),
                                                    b->table->bvh,
                                                    stack.get(),
                                                    stackSize,
                                                    a->table->rows.size(),
                                                    radius);

    hipcub::DeviceScan::ExclusiveSum(cub_tmp_mem.get(), cub_tmp_memsize, pointsInsideLineBuffer.get(),
                                  prefixSumPointsInsideLineBuffer.get(), a->table->rows.size() + 1);
    uint allsize = 0;
    hipMemcpy(&allsize, prefixSumPointsInsideLineBuffer.get() + a->table->rows.size(), sizeof(uint), hipMemcpyDeviceToHost);

    //gpudb::GpuStackAllocator::getInstance().free(cub_tmp_mem);
    cub_tmp_mem.reset();

    auto testedLineNum = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(allsize);
    auto testedResult = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(allsize + 1);
    auto testedPointNum = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(allsize);
    auto testedResultPrefixSum = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(allsize + 1);

    hipcub::DeviceScan::ExclusiveSum(nullptr, cub_tmp_memsize, testedResult.get(), testedResultPrefixSum.get(), allsize + 1);
    cub_tmp_mem = gpudb::GpuStackAllocatorAdditions::allocUnique<uint8_t>(cub_tmp_memsize);

    if (testedLineNum == nullptr || testedResult == nullptr ||
        testedPointNum == nullptr || testedResultPrefixSum == nullptr || cub_tmp_mem == nullptr) {
        return MYERR_STRING("not enough stack memory");
    }

    boxInsideBoxLineKernel2<<<grid, block>>>(thrust::raw_pointer_cast(a->table->rows.data()),
                                                    pointsInsideLineBuffer.get(),
                                                    prefixSumPointsInsideLineBuffer.get(),
                                                    testedLineNum.get(),
                                                    testedPointNum.get(),
                                                    b->table->bvh,
                                                    stack.get(),
                                                    stackSize,
                                                    a->table->rows.size(),
                                                    radius);
    grid = gridConfigure(allsize, block);
    pointInsideLineBufferKernel<<<grid, block>>>(thrust::raw_pointer_cast(a->table->rows.data()),
                                               pointsInsideLineBuffer.get(),
                                               testedLineNum.get(),
                                               testedPointNum.get(),
                                               testedResult.get(),
                                               thrust::raw_pointer_cast(b->table->rows.data()),
                                               allsize, radius);


    hipcub::DeviceScan::ExclusiveSum(cub_tmp_mem.get(), cub_tmp_memsize, testedResult.get(), testedResultPrefixSum.get(), allsize + 1);

    uint totalRowsSelected = 0;
    hipMemcpy(&totalRowsSelected, testedResultPrefixSum.get() + allsize, sizeof(uint), hipMemcpyDeviceToHost);

    auto cpuselectedRows = gpudb::GpuStackAllocatorAdditions::allocUniqueNull<uint>();

    if (totalRowsSelected > 0) {
        auto selectedRows = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(totalRowsSelected);
        cpuselectedRows = StackAllocatorAdditions::allocUnique<uint>(totalRowsSelected);

        if (selectedRows == nullptr || cpuselectedRows == nullptr) {
            return MYERR_STRING("not enough stack memory");
        }

        dim3 grid = gridConfigure(allsize, block);
        selector<<<grid, block>>>(selectedRows.get(), testedPointNum.get(), testedResult.get(), testedResultPrefixSum.get(), allsize);
        hipMemcpy(cpuselectedRows.get(), selectedRows.get(), sizeof(uint) * totalRowsSelected, hipMemcpyDeviceToHost);
    }

    grid = gridConfigure(a->table->rows.size(), block);
    computeResultSize<<<grid, block>>>(pointsInsideLineBuffer.get(), prefixSumPointsInsideLineBuffer.get(), testedResultPrefixSum.get(), a->table->rows.size());
    computeResultSize2<<<grid, block>>>(pointsInsideLineBuffer.get(), a->table->rows.size());

    //gpudb::GpuStackAllocator::getInstance().free(cub_tmp_mem);
    //gpudb::GpuStackAllocator::getInstance().free(testedResultPrefixSum);
    cub_tmp_mem.reset();
    testedResultPrefixSum.reset();

    auto cputestedResultSize = StackAllocatorAdditions::allocUnique<uint>(a->table->rows.size());
    hipMemcpy(cputestedResultSize.get(), pointsInsideLineBuffer.get(), sizeof(uint) * a->table->rows.size(), hipMemcpyDeviceToHost);

    /*uint *cputestedLineNum = StackAllocator::getInstance().alloc<uint>(allsize);
    uint *cputestedResult = StackAllocator::getInstance().alloc<uint>(allsize);
    uint *cputestedPointNum = StackAllocator::getInstance().alloc<uint>(allsize);
    hipMemcpy(cputestedLineNum, testedLineNum, allsize * sizeof(uint), hipMemcpyDeviceToHost);
    hipMemcpy(cputestedResult, testedResult, allsize * sizeof(uint), hipMemcpyDeviceToHost);
    hipMemcpy(cputestedPointNum, testedPointNum, allsize * sizeof(uint), hipMemcpyDeviceToHost);

    for (int i = 0; i < a.table->rows.size(); i++) {
        printf("%d ", cputestedResultSize[i]);
    }
    printf("\n");

    for (uint i = 0; i < allsize; i++) {
        printf("{ point : %d result : %d line : %d} \n", cputestedPointNum[i], cputestedResult[i], cputestedLineNum[i]);
    }*/

    return resultToTempTable1(a, b, "Linebuffer operation result", cpuselectedRows.get(), cputestedResultSize.get());
}

Result<std::unique_ptr<TempTable>, Error<std::string>>
DataBase::polygonxpointPointsInPolygon(std::unique_ptr<TempTable> const &a, std::unique_ptr<TempTable> &b) {
    if (a == nullptr || b == nullptr) {
        return MYERR_STRING("TempTable a or b is nullptr");
    }

    if (!a->isValid() || !b->isValid()) {
        return MYERR_STRING("TempTable a or b is invalid");
    }

    if (a->table == nullptr ||
        b->table == nullptr)
    {
        return MYERR_STRING("a.table  or b.table is nullptr");
    }

    if (a->getSpatialKeyType() != SpatialType::POLYGON ||
        b->getSpatialKeyType() != SpatialType::POINT)
    {
        return MYERR_STRING("a or b type mismatch");
    }

    if (a->table->rows.size() == 0 || b->table->rows.size() == 0) {
        return MYERR_STRING("TempTable a or b has no rows");
    }

    if (!b->table->bvh.isBuilded()) {
        auto boxes = gpudb::GpuStackAllocatorAdditions::allocUnique<gpudb::AABB> (b->table->rows.size());
        if (boxes == nullptr) {
            return MYERR_STRING("not enough gpu stack memory");
        }

        dim3 block(BLOCK_SIZE);
        dim3 grid(gridConfigure(b->table->rows.size(), block));
        buildKeysAABB<<<grid, block>>>(thrust::raw_pointer_cast(b->table->rows.data()), boxes.get(), b->table->rows.size());
        TRY(b->table->bvh.build(boxes.get(), b->table->rows.size()));
    }

    uint stackSize = b->table->bvh.numBVHLevels * 2 + 1;
    auto stack = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(stackSize * a->table->rows.size());
    auto pointsInsidePolygon = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(a->table->rows.size() + 1);
    auto prefixSumPointsInsidePolygon = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(a->table->rows.size() + 1);

    uint64_t cub_tmp_memsize = 0;
    hipcub::DeviceScan::ExclusiveSum(nullptr, cub_tmp_memsize, pointsInsidePolygon.get(), prefixSumPointsInsidePolygon.get(), a->table->rows.size() + 1);
    auto cub_tmp_mem = gpudb::GpuStackAllocatorAdditions::allocUnique<uint8_t>(cub_tmp_memsize);
    auto cpuPointsInsidePolygon = StackAllocatorAdditions::allocUnique<uint>(a->table->rows.size());

    if (pointsInsidePolygon == nullptr ||
        stack == nullptr ||
        cpuPointsInsidePolygon == nullptr ||
        cub_tmp_mem == nullptr ||
        prefixSumPointsInsidePolygon == nullptr) {
        return MYERR_STRING("not enough stack memory");
    }

    dim3 block(BLOCK_SIZE);
    dim3 grid(gridConfigure(a->table->rows.size(), block));
    boxInsideBoxKernel<<<grid, block>>>(thrust::raw_pointer_cast(a->table->rows.data()),
                                                    pointsInsidePolygon.get(),
                                                    b->table->bvh,
                                                    stack.get(),
                                                    stackSize,
                                                    a->table->rows.size());

    hipcub::DeviceScan::ExclusiveSum(cub_tmp_mem.get(), cub_tmp_memsize, pointsInsidePolygon.get(), prefixSumPointsInsidePolygon.get(), a->table->rows.size() + 1);
    uint allsize = 0;
    hipMemcpy(&allsize, prefixSumPointsInsidePolygon.get() + a->table->rows.size(), sizeof(uint), hipMemcpyDeviceToHost);

    cub_tmp_mem.reset(); // освобождаем память

    // номер точки /результат тестирвования/номер полигона
    auto testedPolygonNum = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(allsize);
    auto testedResult = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(allsize + 1);
    auto testedPointNum = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(allsize);
    auto testedResultPrefixSum = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(allsize + 1);


    hipcub::DeviceScan::ExclusiveSum(nullptr, cub_tmp_memsize, testedResult.get(), testedResultPrefixSum.get(), allsize + 1);
    cub_tmp_mem = gpudb::GpuStackAllocatorAdditions::allocUnique<uint8_t>(cub_tmp_memsize);

    if (testedPolygonNum == nullptr
        || testedResult == nullptr
        || testedPointNum == nullptr
        || testedResultPrefixSum == nullptr
        || cub_tmp_mem == nullptr) {
        return MYERR_STRING("not enough stack memory");
    }

    boxInsideBoxKernel2<<<grid, block>>>(thrust::raw_pointer_cast(a->table->rows.data()),
                                                    pointsInsidePolygon.get(),
                                                    prefixSumPointsInsidePolygon.get(),
                                                    testedPolygonNum.get(),
                                                    testedPointNum.get(),
                                                    b->table->bvh,
                                                    stack.get(),
                                                    stackSize,
                                                    a->table->rows.size());
    grid = gridConfigure(allsize, block);
    pointInsidePolygonKernel<<<grid, block>>>(thrust::raw_pointer_cast(a->table->rows.data()),
                                               pointsInsidePolygon.get(),
                                               testedPolygonNum.get(),
                                               testedPointNum.get(),
                                               testedResult.get(),
                                               thrust::raw_pointer_cast(b->table->rows.data()),
                                               allsize);

    hipcub::DeviceScan::ExclusiveSum(cub_tmp_mem.get(), cub_tmp_memsize, testedResult.get(), testedResultPrefixSum.get(), allsize + 1);

    uint totalRowsSelected = 0;
    hipMemcpy(&totalRowsSelected, testedResultPrefixSum.get() + allsize, sizeof(uint), hipMemcpyDeviceToHost);

    std::unique_ptr<uint, void(*)(uint*)> cpuselectedRows(nullptr, StackAllocatorAdditions::free<uint>);
    if (totalRowsSelected > 0) {
        auto selectedRows = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(totalRowsSelected);
        cpuselectedRows = StackAllocatorAdditions::allocUnique<uint>(totalRowsSelected);

        if (selectedRows == nullptr || cpuselectedRows == nullptr) {
            return MYERR_STRING("not enough stack memory");
        }

        dim3 grid = gridConfigure(allsize, block);
        selector<<<grid, block>>>(selectedRows.get(), testedPointNum.get(), testedResult.get(), testedResultPrefixSum.get(), allsize);
        hipMemcpy(cpuselectedRows.get(), selectedRows.get(), sizeof(uint) * totalRowsSelected, hipMemcpyDeviceToHost);
    }

    grid = gridConfigure(a->table->rows.size(), block);
    computeResultSize<<<grid, block>>>(pointsInsidePolygon.get(), prefixSumPointsInsidePolygon.get(), testedResultPrefixSum.get(), a->table->rows.size());
    computeResultSize2<<<grid, block>>>(pointsInsidePolygon.get(), a->table->rows.size());

    cub_tmp_mem.reset();
    testedResultPrefixSum.reset();

    auto cputestedResultSize = StackAllocatorAdditions::allocUnique<uint>(a->table->rows.size());
    hipMemcpy(cputestedResultSize.get(), pointsInsidePolygon.get(), sizeof(uint) * a->table->rows.size(), hipMemcpyDeviceToHost);

    return resultToTempTable1(a, b, "Points inside Polygon operation result", cpuselectedRows.get(), cputestedResultSize.get());
}

__device__
static float mindist1D(float p, float s, float t) {
    if (p < s) {
        return s;
    }
    if (p > t) {
        return t;
    }
    return p;
}

__device__
static float mindist(float2 p, float4 s, float4 t) {
    float2 r;
    r.x = mindist1D(p.x, s.x, t.x);
    r.y = mindist1D(p.y, s.y, t.y);
    return sqr(p.x - r.x) + sqr(p.y - r.y);
}

__device__
static  float minmaxdist(float2 p, float4 s, float4 t) {
    float2 rM, rm;
    rM.x = (2.0f * p.x >= (s.x + t.x))? s.x : t.x;
    rM.y = (2.0f * p.y >= (s.y + t.y))? s.y : t.y;

    rm.x = (2.0f * p.x <= (s.x + t.x))? s.x : t.x;
    rm.y = (2.0f * p.y <= (s.y + t.y))? s.y : t.y;

    float d1 = sqr(p.x - rm.x) + sqr(p.y - rM.y);
    float d2 = sqr(p.y - rm.y) + sqr(p.x - rM.x);

    return min(d1 + 2 * d1 * FLT_EPSILON, d2 + 2 * d2 * FLT_EPSILON);
}

#define NOT_USED 0xFFFFFFFF

__device__ void visitOrder(uint pos,
                           gpudb::HLBVH &bvh,
                           float2 point,
                           Heap<float, uint, uint> &heap,
                           GpuStack<uint2> &st)
{
    float4 bmin1 = bvh.aabbMin[pos];
    float4 bmax1 = bvh.aabbMax[pos];
    float4 bmin2 = bvh.aabbMin[pos + 1];
    float4 bmax2 = bvh.aabbMax[pos + 1];
    float min1 = mindist(point, bmin1, bmax1);
    float min2 = mindist(point, bmin2, bmax2);
    float minmax1 = minmaxdist(point, bmin1, bmax1);
    float minmax2 = minmaxdist(point, bmin2, bmax2);
    uint *memoryRef1 = 0;
    uint *memoryRef2 = 0;

    if (min1 < min2) {
        st.push(make_uint2(pos + 1, NOT_USED));
        memoryRef2 = &st.topRef().y;

        st.push(make_uint2(pos, NOT_USED));
        memoryRef1 = &st.topRef().y;
    } else {
        st.push(make_uint2(pos, NOT_USED));
        memoryRef1 = &st.topRef().y;

        st.push(make_uint2(pos + 1, NOT_USED));
        memoryRef2 = &st.topRef().y;
    }

    if (minmax1 < heap.maxKey()) {
        if (heap.count == heap.cap) {
            heap.extractMax();
        }
        heap.insert(minmax1, -1, memoryRef1);
    }

    if (minmax2 < heap.maxKey()) {
        if (heap.count == heap.cap) {
            heap.extractMax();
        }
        heap.insert(minmax2, -1, memoryRef2);
    }
}

__global__
void knearestNeighbor(gpudb::HLBVH bvh,
                      gpudb::GpuRow **search,
                      gpudb::GpuRow **data,
                      float *heapKeys,
                      uint *heapValues,
                      uint **heapIndexes,
                      uint2 *stack,
                      uint stackSize,
                      uint k,
                      uint workSize)
{
    uint idx = getGlobalIdx3DZXY();
    if (idx >= workSize) {
        return;
    }

    float2 point;
    point.x = ((gpudb::GpuPoint*)search[idx]->spatialPart.key)->p.x;
    point.y = ((gpudb::GpuPoint*)search[idx]->spatialPart.key)->p.y;

    GpuStack<uint2> st(stack + idx * stackSize, stackSize);

    Heap<float, uint, uint> heap(heapKeys + idx *  k, heapValues + idx * k, heapIndexes + idx * k, k);
    heap.count = heap.cap;

    for (int i = 0; i < k; i++) {
        heap.keys[i] = INFINITY;
        heap.values[i] = -1;
        heap.indexes[i] = nullptr;
    }

    visitOrder(0, bvh, point, heap, st);
    while(!st.empty()) {
        uint2 posSt = st.top(); st.pop();
        uint pos = posSt.x;
        uint ref = posSt.y;
        int link = bvh.links[pos];

        float4 bmin1 = bvh.aabbMin[pos];
        float4 bmax1 = bvh.aabbMax[pos];

        if (heap.maxKey() < mindist(point, bmin1, bmax1)) {
            continue;
        }

        if (ref != NOT_USED) {
            heap.deleteKey(ref);
            heap.insert(INFINITY, -1, nullptr);
        }

        if (link == LEAF) {
            for (int i = bvh.ranges[pos].x; i < bvh.ranges[pos].y; i++) {
                uint bvhref = bvh.references[i];
                float2 p = ((gpudb::GpuPoint*)data[bvhref]->spatialPart.key)->p;

                float dist = lenSqr(p, point);
                if (dist < heap.maxKey()) {
                    if (heap.cap == heap.count) {
                        heap.extractMax();
                    }
                    heap.insert(dist, bvhref, nullptr);
                }
            }
        } else {
            visitOrder(link, bvh, point, heap, st);
        }
    }

    while(!heap.empty()) {
        uint heapV = heap.maxValue();
        float heapK = heap.maxKey();
        heap.extractMax();
        heap.values[heap.count] = heapV;
        heap.keys[heap.count] = heapK;
    }
}

Result<std::unique_ptr<TempTable>, Error<std::string>>
DataBase::pointxpointKnearestNeighbor(std::unique_ptr<TempTable> const &a, std::unique_ptr<TempTable> &b, uint k) {
    if (a == nullptr || b == nullptr) {
        return MYERR_STRING("TempTable a or b is nullptr");
    }

    if (!a->isValid() || !b->isValid()) {
        return MYERR_STRING("TempTable a or b is invalid");
    }

    if (a->table == nullptr ||
        b->table == nullptr)
    {
        return MYERR_STRING("a.table  or b.table is nullptr");
    }

    if (a->getSpatialKeyType() != SpatialType::POINT ||
        b->getSpatialKeyType() != SpatialType::POINT)
    {
        return MYERR_STRING("a or b type mismatch");
    }

    if (a->table->rows.size() == 0) {
        return MYERR_STRING("TempTable a has no rows");
    }

    if (b->table->rows.size() < k) {
        return MYERR_STRING("TempTable b has rows.size() less than k");
    }

    if (k == 0) {
        return MYERR_STRING(string_format("k mast be in 0 < %d <= %d", k, b->table->rowsSize.size()));
    }

    if (!b->table->bvh.isBuilded()) {
        auto boxes = gpudb::GpuStackAllocatorAdditions::allocUnique<gpudb::AABB> (b->table->rows.size());
        if (boxes == nullptr) {
            return MYERR_STRING("not enough gpu stack memory");
        }

        dim3 block(BLOCK_SIZE);
        dim3 grid(gridConfigure(b->table->rows.size(), block));
        buildKeysAABB<<<grid, block>>>(thrust::raw_pointer_cast(b->table->rows.data()), boxes.get(), b->table->rows.size());
        TRY(b->table->bvh.build(boxes.get(), b->table->rows.size()));
    }

    uint stackSize = b->table->bvh.numBVHLevels * 2 + 1;
    auto heapKeys = gpudb::GpuStackAllocatorAdditions::allocUnique<float>(k * a->table->rows.size());
    auto heapValues = gpudb::GpuStackAllocatorAdditions::allocUnique<uint>(k * a->table->rows.size());
    auto heapIndexes = gpudb::GpuStackAllocatorAdditions::allocUnique<uint*>(k * a->table->rows.size());
    auto stack = gpudb::GpuStackAllocatorAdditions::allocUnique<uint2>(stackSize * a->table->rows.size());
    auto result = StackAllocatorAdditions::allocUnique<uint>(k * a->table->rows.size());
    if (heapIndexes == nullptr
        || stack == nullptr
        || heapKeys == nullptr
        || heapValues == nullptr
        || result == nullptr) {
        return MYERR_STRING("not enough stack memory");
    }

    dim3 block(BLOCK_SIZE);
    dim3 grid(gridConfigure(a->table->rows.size(), block));
    Timer t;
    t.start();
    knearestNeighbor<<<grid, block>>>(b->table->bvh,
                                      thrust::raw_pointer_cast(a->table->rows.data()),
                                      thrust::raw_pointer_cast(b->table->rows.data()),
                                      heapKeys.get(),
                                      heapValues.get(),
                                      heapIndexes.get(),
                                      stack.get(),
                                      stackSize,
                                      k,
                                      a->table->rows.size());
    gLogWrite(LOG_MESSAGE_TYPE::DEBUG, "k nearest neighbor in %d ms", t.elapsedMillisecondsU64());
    hipMemcpy(result.get(), heapValues.get(), sizeof(uint) * k * a->table->rows.size(), hipMemcpyDeviceToHost);
    auto sizes = StackAllocatorAdditions::allocUnique<uint>(a->table->rows.size());

    if (sizes == nullptr) {
        return MYERR_STRING("not enough stack memory");
    }

    for (uint i = 0; i < a->table->rows.size(); i++) {
        sizes.get()[i] = k;
    }

    return resultToTempTable1(a, b, "k nearest neighbor",result.get(), sizes.get());
}
